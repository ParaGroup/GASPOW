/**
  Testing a templated kernel
*/

#ifndef TEST_TEMPLATE_CU
#define TEST_TEMPLATE_CU
#include <iostream>
#include <vector>
#include <functional>
using namespace std;

// struct of the input data type
typedef struct input_struct {
	int key;
	int id;
	long int value;
} input_t;

// struct of the output data type
typedef struct output_struct {
	int key;
	int id;
	long int value;
} output_t;

//struct of the support data structure to correctly invoke the function passed by the user

template<typename T_IN, typename T_OUT>
struct kernel_support_t{
	T_IN *data;	//pointer to the beginning of data
	T_OUT *res; //pointer to result;
	unsigned long size; //length of data
};

__global__ void ff()
{
	printf("Hello %d\n",3);
}

template<typename T_IN, typename T_OUT, typename T_F>
__global__ void kernel(kernel_support_t<T_IN,T_OUT> *ks, T_F fun)
{
	//tmp cast
	//input_t* d=static_cast<input_t *>(data);
	//output_t* r=static_cast<output_t*>(res);
	//r->value=0;
	//for(int i=0;i<size;i++)
	//	r->value+=d[i].value;


	fun(ks->data,ks->res,ks->size);
}

template<typename T, typename CUDA_F>
void launch_kernel(vector<T> v, int size, CUDA_F lambda)
{
	//allocate data for the gpu

	input_t *gpu_data;
	output_t *gpu_res, *host_res;
	kernel_support_t<input_t,output_t> *kt_gpu,*kt;

	hipMalloc((kernel_support_t<input_t,output_t> **)&kt_gpu,sizeof(kernel_support_t<input_t,output_t>));
	hipMalloc((input_t **)&gpu_data,size*sizeof(input_t));
	hipMalloc((output_t **)&gpu_res,sizeof(output_t));
	host_res=(output_t*)malloc(sizeof(host_res));
	kt=(kernel_support_t<input_t,output_t>*)malloc(sizeof(kernel_support_t<input_t,output_t>));

	//fill kernel support info
	kt->data=gpu_data;
	kt->res=gpu_res;
	kt->size=size;

	//copy data to gpu
	hipMemcpy(kt_gpu,kt,sizeof(kernel_support_t<input_t,output_t>),hipMemcpyHostToDevice);
	hipMemcpy(gpu_data,v.data(),sizeof(input_t)*size,hipMemcpyHostToDevice);

	//kernel definition

	kernel<input_t,output_t><<<1,1>>>(kt_gpu,lambda);


	//copy result back
	hipMemcpy(host_res,gpu_res,sizeof(output_t),hipMemcpyDeviceToHost);


	cout << "Result "<<host_res->value<< ", expected "<< (size*(size-1))/2<<endl;
	//cudaDeviceSynchronize();

	hipFree(gpu_data);
	hipFree(gpu_res);
	hipFree(kt_gpu);

}

int main(int argc, char *argv[])
{
	const int size = 500;
	//allocate data....this will be the window contet
	vector<input_t> v;
	input_t tmp;
	tmp.key=1;
	for(int i=0;i<size;i++)
	{
		tmp.id=i;
		tmp.value=i;
		v.push_back(tmp);
	}


	auto lambda = [=] __device__ (input_t *d, output_t* r, int size) {
		r->value=0;
		for(int i=0;i<size;i++)
			r->value+=d[i].value;
	  };


	launch_kernel<input_t>(v,size,lambda);

}


#endif // TEST_TEMPLATE_CU
